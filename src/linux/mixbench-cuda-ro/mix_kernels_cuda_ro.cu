#include "hip/hip_runtime.h"
/**
 * mix_kernels_cuda_ro.cu: This file is part of the mixbench GPU micro-benchmark suite.
 *
 * Contact: Elias Konstantinidis <ekondis@gmail.com>
 **/

#include <stdio.h>
#include <hip/hip_math_constants.h>
#include "lcutil.h"
#include "mix_kernels_cuda.h"

#define ELEMENTS_PER_THREAD (8)
#define FUSION_DEGREE (4)

BenchType datatype = BENCH_FLOAT;
int secs = 5;

template <unsigned int granularity, unsigned int fusion_degree>
__global__ void benchmark_int(int seed, int *g_data, int compute_iterations){
	const unsigned int blockSize = blockDim.x;
	const int stride = blockSize;
	int idx = blockIdx.x*blockSize*granularity + threadIdx.x;
	const int big_stride = gridDim.x*blockSize*granularity;

	int tmps[granularity];
	for(int k=0; k<fusion_degree; k++){
		#pragma unroll
		for(int j=0; j<granularity; j++){
			// Load elements (memory intensive part)
			tmps[j] = g_data[idx+j*stride+k*big_stride];
			// Perform computations (compute intensive part)
			for(int i=0; i<compute_iterations; i++){
				tmps[j] = tmps[j]*tmps[j]+seed;//tmps[(j+granularity/2)%granularity];
			}
		}
		// Multiply add reduction
		int sum = (int)0;
		#pragma unroll
		for(int j=0; j<granularity; j+=2)
			sum += tmps[j]*tmps[j+1];
		// Dummy code
		if( sum==(int)-1 ) // Designed so it never executes
			g_data[idx+k*big_stride] = sum;
	}
}

template <unsigned int granularity, unsigned int fusion_degree>
__global__ void benchmark_float(float seed, float *g_data, int compute_iterations){
	const unsigned int blockSize = blockDim.x;
	const int stride = blockSize;
	int idx = blockIdx.x*blockSize*granularity + threadIdx.x;
	const int big_stride = gridDim.x*blockSize*granularity;

	float tmps[granularity];
	for(int k=0; k<fusion_degree; k++){
		#pragma unroll
		for(int j=0; j<granularity; j++){
			// Load elements (memory intensive part)
			tmps[j] = g_data[idx+j*stride+k*big_stride];
			// Perform computations (compute intensive part)
			for(int i=0; i<compute_iterations; i++){
				tmps[j] = tmps[j]*tmps[j]+seed;//tmps[(j+granularity/2)%granularity];
			}
		}
		// Multiply add reduction
		float sum = (float)0;
		#pragma unroll
		for(int j=0; j<granularity; j+=2)
			sum += tmps[j]*tmps[j+1];
		// Dummy code
		if( sum==(float)-1 ) // Designed so it never executes
			g_data[idx+k*big_stride] = sum;
	}
}

template <unsigned int granularity, unsigned int fusion_degree>
__global__ void benchmark_double(double seed, double *g_data, int compute_iterations){
	const unsigned int blockSize = blockDim.x;
	const int stride = blockSize;
	int idx = blockIdx.x*blockSize*granularity + threadIdx.x;
	const int big_stride = gridDim.x*blockSize*granularity;

	double tmps[granularity];
	for(int k=0; k<fusion_degree; k++){
		#pragma unroll
		for(int j=0; j<granularity; j++){
			// Load elements (memory intensive part)
			tmps[j] = g_data[idx+j*stride+k*big_stride];
			// Perform computations (compute intensive part)
			for(int i=0; i<compute_iterations; i++){
				tmps[j] = tmps[j]*tmps[j]+seed;//tmps[(j+granularity/2)%granularity];
			}
		}
		// Multiply add reduction
		double sum = (double)0;
		#pragma unroll
		for(int j=0; j<granularity; j+=2)
			sum += tmps[j]*tmps[j+1];
		// Dummy code
		if( sum==(double)-1 ) // Designed so it never executes
			g_data[idx+k*big_stride] = sum;
	}
}

template <class T, unsigned int granularity, unsigned int fusion_degree>
void benchmark_func(dim3 dimGrid, dim3 dimBlock, T seed, T *g_data, int compute_iterations){
    if constexpr(std::is_integral_v<T>)
	benchmark_int<granularity, fusion_degree><<<dimGrid, dimBlock>>>(seed, g_data, compute_iterations);
    else if constexpr (sizeof(T) == 4)
	benchmark_float<granularity, fusion_degree><<<dimGrid, dimBlock>>>(seed, g_data, compute_iterations);
    else
	benchmark_double<granularity, fusion_degree><<<dimGrid, dimBlock>>>(seed, g_data, compute_iterations);

}


void initializeEvents(hipEvent_t *start, hipEvent_t *stop){
	CUDA_SAFE_CALL( hipEventCreate(start) );
	CUDA_SAFE_CALL( hipEventCreate(stop) );
	CUDA_SAFE_CALL( hipEventRecord(*start, 0) );
}

float finalizeEvents(hipEvent_t start, hipEvent_t stop){
	CUDA_SAFE_CALL( hipGetLastError() );
	CUDA_SAFE_CALL( hipEventRecord(stop, 0) );
	CUDA_SAFE_CALL( hipEventSynchronize(stop) );
	float kernel_time;
	CUDA_SAFE_CALL( hipEventElapsedTime(&kernel_time, start, stop) );
	CUDA_SAFE_CALL( hipEventDestroy(start) );
	CUDA_SAFE_CALL( hipEventDestroy(stop) );
	return kernel_time;
}

void runbench_warmup(double *cd, long size){
	const long reduced_grid_size = size/(ELEMENTS_PER_THREAD)/128;
	const int BLOCK_SIZE = 256;
	const int TOTAL_REDUCED_BLOCKS = reduced_grid_size/BLOCK_SIZE;

	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimReducedGrid(TOTAL_REDUCED_BLOCKS, 1, 1);

	benchmark_func<double, ELEMENTS_PER_THREAD, FUSION_DEGREE >(dimReducedGrid, dimBlock, (double)1, (double*)cd, 0);
	CUDA_SAFE_CALL( hipGetLastError() );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

int out_config = 1;

void runbench(double *cd, long size, int compute_iterations){
	const long compute_grid_size = size/ELEMENTS_PER_THREAD/FUSION_DEGREE;
	const int BLOCK_SIZE = 256;
	const int TOTAL_BLOCKS = compute_grid_size/BLOCK_SIZE;
	const long long computations = (ELEMENTS_PER_THREAD*(long long)compute_grid_size+(2*ELEMENTS_PER_THREAD*compute_iterations)*(long long)compute_grid_size)*FUSION_DEGREE;
	const long long memoryoperations = size;

	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(TOTAL_BLOCKS, 1, 1);
	hipEvent_t start, stop;

    float kernel_time_mad = 0.0;
    unsigned int size_of_data = sizeof(float);
    switch (datatype)
    {
        default:
        case BENCH_FLOAT:{
	        initializeEvents(&start, &stop);
	        benchmark_func< float, ELEMENTS_PER_THREAD, FUSION_DEGREE>(dimGrid, dimBlock, 1.0f, (float*)cd, compute_iterations);
	        kernel_time_mad = finalizeEvents(start, stop);
            size_of_data = sizeof(float);
            break;
        }

        case BENCH_DOUBLE:{
	        initializeEvents(&start, &stop);
	        benchmark_func< double, ELEMENTS_PER_THREAD, FUSION_DEGREE>(dimGrid, dimBlock, 1.0, cd, compute_iterations);
	        kernel_time_mad = finalizeEvents(start, stop);
            size_of_data = sizeof(double);
            break;
        }

        case BENCH_INT:{
	        initializeEvents(&start, &stop);
	        benchmark_func< int, ELEMENTS_PER_THREAD, FUSION_DEGREE>(dimGrid, dimBlock, 1, (int*)cd, compute_iterations);
	        kernel_time_mad = finalizeEvents(start, stop);
            size_of_data = sizeof(int);
            break;
        }
    }


	// printf("         %4d,   %8.3f,%8.2f,%8.2f,%7.2f,   %8.3f,%8.2f,%8.2f,%7.2f,  %8.3f,%8.2f,%8.2f,%7.2f\n",
	// 	compute_iterations,
	// 	((double)computations)/((double)memoryoperations*sizeof(float)),
	// 	kernel_time_mad_sp,
	// 	((double)computations)/kernel_time_mad_sp*1000./(double)(1000*1000*1000),
	// 	((double)memoryoperations*sizeof(float))/kernel_time_mad_sp*1000./(1000.*1000.*1000.),
	// 	((double)computations)/((double)memoryoperations*sizeof(double)),
	// 	kernel_time_mad_dp,
	// 	((double)computations)/kernel_time_mad_dp*1000./(double)(1000*1000*1000),
	// 	((double)memoryoperations*sizeof(double))/kernel_time_mad_dp*1000./(1000.*1000.*1000.),
	// 	((double)computations)/((double)memoryoperations*sizeof(int)),
	// 	kernel_time_mad_int,
	// 	((double)computations)/kernel_time_mad_int*1000./(double)(1000*1000*1000),
	// 	((double)memoryoperations*sizeof(int))/kernel_time_mad_int*1000./(1000.*1000.*1000.) );
	printf("         %4d,   %8.3f,%8.2f,%8.2f,%7.2f \n",
		compute_iterations,
		((double)computations)/((double)memoryoperations*sizeof(float)),
		kernel_time_mad,
	    ((double)computations)/kernel_time_mad*1000./(double)(1000*1000*1000),
	    ((double)memoryoperations*size_of_data)/kernel_time_mad*1000./(1000.*1000.*1000.));

    // collect power data
    int iters = int((double)secs * 1000 / kernel_time_mad);
    printf("Adjust %d iterations to achieve time duration %d.\n", iters, secs);

	initializeEvents(&start, &stop);
    for (int i = 0 ; i < iters; i++)
        switch (datatype)
        {
            default:
            case BENCH_FLOAT:{
	            benchmark_func< float, ELEMENTS_PER_THREAD, FUSION_DEGREE>(dimGrid, dimBlock, 1.0f, (float*)cd, compute_iterations);
                break;
            }

            case BENCH_DOUBLE:{
	            benchmark_func< double, ELEMENTS_PER_THREAD, FUSION_DEGREE>(dimGrid, dimBlock, 1.0, cd, compute_iterations);
                break;
            }

            case BENCH_INT:{
	            benchmark_func< int, ELEMENTS_PER_THREAD, FUSION_DEGREE>(dimGrid, dimBlock, 1, (int*)cd, compute_iterations);
                break;
            }
        }

    float avg_msec = finalizeEvents(start, stop) / iters;
    printf("benchmark_func() iterated %d, average time is %f msec\n", iters, avg_msec);

}

extern "C" void mixbenchGPU(double *c, long size, int compute_iterations, BenchType bt, int s){
	const char *benchtype = "compute with global memory (block strided)";

	printf("Trade-off type:       %s\n", benchtype);
	printf("Elements per thread:  %d\n", ELEMENTS_PER_THREAD);
	printf("Thread fusion degree: %d\n", FUSION_DEGREE);
	double *cd;
    datatype = bt;
    secs = s;

	CUDA_SAFE_CALL( hipMalloc((void**)&cd, size*sizeof(double)) );

	// Copy data to device memory
	CUDA_SAFE_CALL( hipMemset(cd, 0, size*sizeof(double)) );  // initialize to zeros

	// Synchronize in order to wait for memory operations to finish
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	printf("---------------------------------------------------------- CSV data ----------------------------------------------------------\n");
	// printf("Experiment ID, Single Precision ops,,,,              Double precision ops,,,,              Integer operations,,, \n");
	// printf("Compute iters, Flops/byte, ex.time,  GFLOPS, GB/sec, Flops/byte, ex.time,  GFLOPS, GB/sec, Iops/byte, ex.time,   GIOPS, GB/sec\n");
    if (datatype == BENCH_FLOAT)
	    printf("Experiment ID, Single Precision ops,,,,              \n");
    if (datatype == BENCH_DOUBLE)
	    printf("Experiment ID, Double Precision ops,,,,              \n");
    if (datatype == BENCH_INT)
	    printf("Experiment ID, Int Precision ops,,,,              \n");
	printf("Compute iters, Flops/byte, ex.time,  GFLOPS, GB/sec  \n");

	// runbench_warmup(cd, size);

	runbench(cd, size, compute_iterations); // 0~256

	printf("------------------------------------------------------------------------------------------------------------------------------\n");

	// Copy results back to host memory
	CUDA_SAFE_CALL( hipMemcpy(c, cd, size*sizeof(double), hipMemcpyDeviceToHost) );

	CUDA_SAFE_CALL( hipFree(cd) );

	CUDA_SAFE_CALL( hipDeviceReset() );
}
