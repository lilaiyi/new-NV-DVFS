/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Computation of eigenvalues of a small symmetric, tridiagonal matrix */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
#include "structs.h"
#include "matlab.h"

// includes, kernels
#include "bisect_kernel_small.cuh"

// includes, file
#include "bisect_small.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Determine eigenvalues for matrices smaller than MAX_SMALL_MATRIX
//! @param TimingIterations  number of iterations for timing
//! @param  input  handles to input data of kernel
//! @param  result handles to result of kernel
//! @param  mat_size  matrix size
//! @param  lg  lower limit of Gerschgorin interval
//! @param  ug  upper limit of Gerschgorin interval
//! @param  precision  desired precision of eigenvalues
//! @param  iterations  number of iterations for timing
////////////////////////////////////////////////////////////////////////////////
void
computeEigenvaluesSmallMatrix(const InputData &input, ResultDataSmall &result,
                              const unsigned int mat_size,
                              const float lg, const float ug,
                              const float precision,
							  const unsigned int iterations, const int secs, const int timeRestrict)
{
	dim3  blocks(1, 1, 1);
	dim3  threads(MAX_THREADS_BLOCK_SMALL_MATRIX, 1, 1);

	hipEvent_t start, stop;
	// Record the start event
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	double totalTime = 0.0;
	double averMsecs = 0.0;
	int c = 0;
	int nIter = iterations;

	{
		for (int iter = -20; iter < nIter; iter++)
		{
			// Run kernel and record the time
			checkCudaErrors(hipEventRecord(start, NULL));

			bisectKernel << < blocks, threads >> >(input.g_a, input.g_b, mat_size,
				result.g_left, result.g_right,
				result.g_left_count,
				result.g_right_count,
				lg, ug, 0, mat_size,
				precision
				);

			hipDeviceSynchronize();
			checkCudaErrors(hipEventRecord(stop, NULL));

			// Wait for the stop event to complete
			checkCudaErrors(hipEventSynchronize(stop));
			float msecTotal = 0.0f;
			checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

			totalTime += msecTotal;
			c++;

			//iter == -1 -- warmup iteration
			if (iter == -1)
			{
				checkCudaErrors(hipDeviceSynchronize());
				//sdkResetTimer(&hTimer);
				//sdkStartTimer(&hTimer);
				if (timeRestrict){
					nIter = int(double(secs * 1000) / msecTotal);
					printf("Adjust Iters to %d for meeting time requirement %d secs.\n", nIter, secs);
				}
				totalTime = 0.0;
				c = 0;
			}


		}

	}

	checkCudaErrors(hipDeviceSynchronize());
	//sdkStopTimer(&hTimer);
	averMsecs = totalTime / nIter;

	printf("iterated %d, average time is %f msec.\n", nIter, averMsecs);

}

////////////////////////////////////////////////////////////////////////////////
//! Initialize variables and memory for the result for small matrices
//! @param result  handles to the necessary memory
//! @param  mat_size  matrix_size
////////////////////////////////////////////////////////////////////////////////
void
initResultSmallMatrix(ResultDataSmall &result, const unsigned int mat_size)
{

    result.mat_size_f = sizeof(float) * mat_size;
    result.mat_size_ui = sizeof(unsigned int) * mat_size;

    result.eigenvalues = (float *) malloc(result.mat_size_f);

    // helper variables
    result.zero_f = (float *) malloc(result.mat_size_f);
    result.zero_ui = (unsigned int *) malloc(result.mat_size_ui);

    for (unsigned int i = 0; i < mat_size; ++i)
    {

        result.zero_f[i] = 0.0f;
        result.zero_ui[i] = 0;

        result.eigenvalues[i] = 0.0f;
    }

    checkCudaErrors(hipMalloc((void **) &result.g_left, result.mat_size_f));
    checkCudaErrors(hipMalloc((void **) &result.g_right, result.mat_size_f));

    checkCudaErrors(hipMalloc((void **) &result.g_left_count,
                               result.mat_size_ui));
    checkCudaErrors(hipMalloc((void **) &result.g_right_count,
                               result.mat_size_ui));

    // initialize result memory
    checkCudaErrors(hipMemcpy(result.g_left, result.zero_f, result.mat_size_f,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(result.g_right, result.zero_f, result.mat_size_f,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(result.g_right_count, result.zero_ui,
                               result.mat_size_ui,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(result.g_left_count, result.zero_ui,
                               result.mat_size_ui,
                               hipMemcpyHostToDevice));
}

////////////////////////////////////////////////////////////////////////////////
//! Cleanup memory and variables for result for small matrices
//! @param  result  handle to variables
////////////////////////////////////////////////////////////////////////////////
void
cleanupResultSmallMatrix(ResultDataSmall &result)
{

    freePtr(result.eigenvalues);
    freePtr(result.zero_f);
    freePtr(result.zero_ui);

    checkCudaErrors(hipFree(result.g_left));
    checkCudaErrors(hipFree(result.g_right));
    checkCudaErrors(hipFree(result.g_left_count));
    checkCudaErrors(hipFree(result.g_right_count));
}

////////////////////////////////////////////////////////////////////////////////
//! Process the result obtained on the device, that is transfer to host and
//! perform basic sanity checking
//! @param  input  handles to input data
//! @param  result  handles to result data
//! @param  mat_size   matrix size
//! @param  filename  output filename
////////////////////////////////////////////////////////////////////////////////
void
processResultSmallMatrix(const InputData &input, const ResultDataSmall &result,
                         const unsigned int mat_size,
                         const char *filename)
{

    const unsigned int mat_size_f = sizeof(float) * mat_size;
    const unsigned int mat_size_ui = sizeof(unsigned int) * mat_size;

    // copy data back to host
    float *left = (float *) malloc(mat_size_f);
    unsigned int *left_count = (unsigned int *) malloc(mat_size_ui);

    checkCudaErrors(hipMemcpy(left, result.g_left, mat_size_f,
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(left_count, result.g_left_count, mat_size_ui,
                               hipMemcpyDeviceToHost));

    float *eigenvalues = (float *) malloc(mat_size_f);

    for (unsigned int i = 0; i < mat_size; ++i)
    {
        eigenvalues[left_count[i]] = left[i];
    }

    // save result in matlab format
    writeTridiagSymMatlab(filename, input.a, input.b+1, eigenvalues, mat_size);

    freePtr(left);
    freePtr(left_count);
    freePtr(eigenvalues);
}
