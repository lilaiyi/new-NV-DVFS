#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample calculates scalar products of a
 * given set of input vector pairs
 */



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProdCPU(
    float *h_C,
    float *h_A,
    float *h_B,
    int vectorN,
    int elementN
);



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cuh"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

//Total number of input vector pairs; arbitrary
int VECTOR_N = 256;
//Number of elements per vector; arbitrary,
//but strongly preferred to be a multiple of warp size
//to meet memory coalescing constraints
const int ELEMENT_N = 4096;
//Total number of data elements
int    DATA_N = VECTOR_N * ELEMENT_N;
int   DATA_SZ = DATA_N * sizeof(float);
int RESULT_SZ = VECTOR_N  * sizeof(float);



///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int nIter = 100;
int secs = 180;
bool timeRestrict = false;

int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
    float *d_A, *d_B, *d_C;
    double delta, ref, sum_delta, sum_ref, L1norm;
    StopWatchInterface *hTimer = NULL;
    int i;

    printf("%s Starting...\n\n", argv[0]);

	// array size
	if (checkCmdLineFlag(argc, (const char **)argv, "N"))
	{
		VECTOR_N = getCmdLineArgumentInt(argc, (const char **)argv, "N") * 1024;
		DATA_N = VECTOR_N * ELEMENT_N;

		DATA_SZ = DATA_N * sizeof(float);
		RESULT_SZ = VECTOR_N  * sizeof(float);
	}

	// Iteration count
	if (checkCmdLineFlag(argc, (const char **)argv, "iters"))
	{
		nIter = getCmdLineArgumentInt(argc, (const char **)argv, "iters");
	}

	// Power Running Time
	if (checkCmdLineFlag(argc, (const char **)argv, "secs"))
	{
		secs = getCmdLineArgumentInt(argc, (const char **)argv, "secs");
		timeRestrict = true;
	}

	int dev = findCudaDevice(argc, (const char **)argv);

	if (dev == -1)
	{
		return EXIT_FAILURE;
	}

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory.\n");
    h_A     = (float *)malloc(DATA_SZ);
    h_B     = (float *)malloc(DATA_SZ);
    h_C_CPU = (float *)malloc(RESULT_SZ);
    h_C_GPU = (float *)malloc(RESULT_SZ);

    printf("...allocating GPU memory.\n");
    checkCudaErrors(hipMalloc((void **)&d_A, DATA_SZ));
    checkCudaErrors(hipMalloc((void **)&d_B, DATA_SZ));
    checkCudaErrors(hipMalloc((void **)&d_C, RESULT_SZ));

    printf("...generating input data in CPU mem.\n");
    srand(123);

    //Generating input data on CPU
    for (i = 0; i < DATA_N; i++)
    {
        h_A[i] = RandFloat(0.0f, 1.0f);
        h_B[i] = RandFloat(0.0f, 1.0f);
    }

    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    checkCudaErrors(hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice));
    printf("Data init done.\n");


    printf("Executing GPU kernel...\n");
    checkCudaErrors(hipDeviceSynchronize());
	
	hipEvent_t start, stop;
	// Record the start event
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	double totalTime = 0.0;

	int k = -1;
	while (k < nIter)
	{
		// Run kernel and record the time
		checkCudaErrors(hipEventRecord(start, NULL));

		scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);

		hipDeviceSynchronize();

		checkCudaErrors(hipEventRecord(stop, NULL));

		// Wait for the stop event to complete
		checkCudaErrors(hipEventSynchronize(stop));
		float msecTotal = 0.0f;
		checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

		//iter == -1 -- warmup iteration
		if (k == -1)
		{
			checkCudaErrors(hipDeviceSynchronize());
			//sdkResetTimer(&hTimer);
			//sdkStartTimer(&hTimer);
			if (timeRestrict){
				nIter = secs / (msecTotal * 0.001);
				printf("Adjust Iters to %d for meeting time requirement %d secs.\n", nIter, secs);
			}
		}
		else
		{
			totalTime += msecTotal;
		}

		k++;

	}

    getLastCudaError("scalarProdGPU() execution failed\n");
    checkCudaErrors(hipDeviceSynchronize());
    //sdkStopTimer(&hTimer);
    printf("Total   Kernel Time scalarProdGPU() time: %f msec\n", totalTime);
    printf("Average Kernel Time scalarProdGPU() time: %f msec\n", totalTime / (double)nIter);
    printf("\n");

    printf("Reading back GPU result...\n");
    //Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost));


    printf("Checking GPU results...\n");
    printf("..running CPU scalar product calculation\n");
    scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

    printf("...comparing the results\n");
    //Calculate max absolute difference and L1 distance
    //between CPU and GPU results
    sum_delta = 0;
    sum_ref   = 0;

    for (i = 0; i < VECTOR_N; i++)
    {
        delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
        ref   = h_C_CPU[i];
        sum_delta += delta;
        sum_ref   += ref;
    }

    L1norm = sum_delta / sum_ref;

    printf("Shutting down...\n");
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_A));
    free(h_C_GPU);
    free(h_C_CPU);
    free(h_B);
    free(h_A);
    sdkDeleteTimer(&hTimer);

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
    printf("L1 error: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "Test passed\n" : "Test failed!\n");
    exit(L1norm < 1e-6 ? EXIT_SUCCESS : EXIT_FAILURE);
}
