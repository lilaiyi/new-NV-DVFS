#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 // halo width along one direction when advancing to the next iteration

// #define BENCH_PRINT

void run(int argc, char** argv);

int rows, cols;
int* data;
int** wall;
int* result;
#define M_SEED 9
int pyramid_height;
int nIter = 100;
int secs = 180;
bool timeRestrict = false;

//#define BENCH_PRINT


void
init(int argc, char** argv)
{
	if(argc>=4){

		cols = atoi(argv[1]);

		rows = atoi(argv[2]);

                pyramid_height=atoi(argv[3]);
	}else{
                printf("Usage: dynproc row_len col_len pyramid_height\n");
                exit(0);
        }
	data = new int[rows*cols];

	wall = new int*[rows];

	for(int n=0; n<rows; n++)

		wall[n]=data+cols*n;

	result = new int[cols];

	

	int seed = M_SEED;

	srand(seed);



	for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            wall[i][j] = rand() % 10;

        }

    }

#ifdef BENCH_PRINT

    for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            printf("%d ",wall[i][j]) ;

        }

        printf("\n") ;

    }

#endif
}

void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel(
                int iteration, 
                int *gpuWall,
                int *gpuSrc,
                int *gpuResults,
                int cols, 
                int rows,
                int startStep,
                int border)
{

        __shared__ int prev[BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx=threadIdx.x;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkX = small_block_cols*bx-border;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int xidx = blkX+tx;
       
        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

        int W = tx-1;
        int E = tx+1;
        
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  isValid){
                  computed = true;
                  int left = prev[W];
                  int up = prev[tx];
                  int right = prev[E];
                  int shortest = MIN(left, up);
                  shortest = MIN(shortest, right);
                  int index = cols*(startStep+i)+xidx;
                  result[tx] = shortest + gpuWall[index];
	
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                prev[tx]= result[tx];
	    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          gpuResults[xidx]=result[tx];		
      }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(blockCols);  
	
    int src = 1, dst = 0;
	hipEvent_t start, stop;
	// Record the start event
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	double totalTime = 0.0;

	int k = -1;
	double gpuTime = 0;
	while (k < nIter)
	{
		// Run kernel and record the time
		checkCudaErrors(hipEventRecord(start, NULL));

		dynproc_kernel << <dimGrid, dimBlock >> >(
			MIN(pyramid_height, rows - 1),
			gpuWall, gpuResult[src], gpuResult[dst],
			cols, rows, 0, borderCols);

		hipDeviceSynchronize();

		checkCudaErrors(hipEventRecord(stop, NULL));

		// Wait for the stop event to complete
		checkCudaErrors(hipEventSynchronize(stop));
		float msecTotal = 0.0f;
		checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

		//iter == -1 -- warmup iteration
		if (k == -1)
		{
			checkCudaErrors(hipDeviceSynchronize());
			//sdkResetTimer(&hTimer);
			//sdkStartTimer(&hTimer);
			if (timeRestrict){
				nIter = secs / (msecTotal * 0.001);
				printf("Adjust Iters to %d for meeting time requirement %d secs.\n", nIter, secs);
			}
		}
		else
		{
			totalTime += msecTotal;
		}

		k++;

	}

	hipDeviceSynchronize();

	//sdkStopTimer(&hTimer);
	printf("iterated %d, average time is %f msec.\n", nIter, totalTime / nIter);

    return dst;
}

int main(int argc, char** argv)
{
	findCudaDevice(argc, (const char **)argv);
	// Power Running Time
	if (checkCmdLineFlag(argc, (const char **)argv, "secs"))
	{
		secs = getCmdLineArgumentInt(argc, (const char **)argv, "secs");
		timeRestrict = true;
	}
	// Iteration count
	if (checkCmdLineFlag(argc, (const char **)argv, "iters"))
	{
		nIter = getCmdLineArgumentInt(argc, (const char **)argv, "iters");
	}

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);


    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, \
	 pyramid_height, blockCols, borderCols);

    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);


#ifdef BENCH_PRINT

    for (int i = 0; i < cols; i++)

            printf("%d ",data[i]) ;

    printf("\n") ;

    for (int i = 0; i < cols; i++)

            printf("%d ",result[i]) ;

    printf("\n") ;

#endif


    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;

}

